#include "hip/hip_runtime.h"
#ifndef _2DCONVOLUTION_KERNEL_H_
#define _2DCONVOLUTION_KERNEL_H_

#include <stdio.h>
#include "2Dconvolution.h"

// Matrix multiplication kernel thread specification
__global__ void ConvolutionKernel(Matrix N, Matrix P)
{
    // Allocate Block_size x Block_size shared memory space
    __shared__ float N_s[BLOCK_SIZE][BLOCK_SIZE];

    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int row_o = ty + blockIdx.y * TILE_SIZE;
    int col_o = tx + blockIdx.x * TILE_SIZE;


    // shift the indecies to get the correct input indecies 
    int row_i = row_o - KERNEL_SIZE/2;
    int col_i = col_o - KERNEL_SIZE/2;

    // Load all the elements needed for the tile 
    if(row_i >= 0 && row_i < N.height && col_i >= 0 && col_i < N.width)
        N_s[ty][tx] = N.elements[row_i*N.width + col_i];
    else
        N_s[ty][tx] = 0.0f;

    
    __syncthreads();

    // only the output threads compute and store the results
    if(tx < TILE_SIZE && ty < TILE_SIZE){
        float pValue = 0.0f;
        for(int y=0; y<KERNEL_SIZE; y++)
            for(int x=0; x<KERNEL_SIZE; x++)
                pValue += Mc[y*KERNEL_SIZE + x] * N_s[y+ty][x+tx];
    
        if(row_o < P.height && col_o < P.width)
            P.elements[row_o*P.width + col_o] = pValue;
    }

}

#endif // #ifndef _2DCONVOLUTION_KERNEL_H_
